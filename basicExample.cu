#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "math.h"
#include <math.h>

#include ""

#include <stdio.h>

struct DeviceFunctionStruct {
    void (*deviceFunctionPointer)(int);
};

__global__ void myKernel(DeviceFunctionStruct* deviceFunctionStruct) {
    if(threadIdx.x == 0)
        printf("in kernel\n");
    void (*deviceFunction)(int) = deviceFunctionStruct->deviceFunctionPointer;
    deviceFunction(42);
    
}

__device__ void myDeviceFunction(int arg) {
    printf("myDeviceFunction(%d)\n", arg);
}

int main() {
    DeviceFunctionStruct* deviceFunctionStruct;
    hipMalloc((void**)&deviceFunctionStruct, sizeof(DeviceFunctionStruct));
    hipMemcpyFromSymbol(&deviceFunctionStruct->deviceFunctionPointer, HIP_SYMBOL(myDeviceFunction), sizeof(void*));

    myKernel<<<1, 1>>>(deviceFunctionStruct);
    hipDeviceSynchronize();

    hipFree(deviceFunctionStruct);
    return 0;
}
