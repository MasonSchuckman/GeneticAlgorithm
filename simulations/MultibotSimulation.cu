#include "hip/hip_runtime.h"
#include "MultibotSimulation.cuh"
#include <hiprand/hiprand_kernel.h>
#include <random>

// NOTE: this must be present in every derived simulation!
extern __constant__ SimConfig config_d;

__constant__ float MAX_SPEED2 = 50.0f;
__constant__ float MAX_ACCEL = 10.00f;

#define degrees 90.0f
#define ROTATION_ANGLE degrees * 3.141592654f / 180.0f // 90 degrees

__host__ void MultibotSimulation::getStartingParams(float *startingParams)
{
    static int iterationsCompleted = 0;
    //printf("iters completed = %d\n", iterationsCompleted);
    

    // get random target coordinates
    int minPos = -2;
    int maxPos = 2;
    std::random_device rd;                                 // obtain a random seed from hardware
    std::mt19937 eng(rd());                                // seed the generator
    std::uniform_int_distribution<> distr(minPos, maxPos); // define the range
    float targetX = distr(eng);
    float targetY = distr(eng);

    // random starting pos
    float startingX = distr(eng);
    float startingY = distr(eng);

    double r = 5.0 + iterationsCompleted / 10;                // radius of circle
    double angle = ((double)rand() / RAND_MAX) * 2 * 3.14159; // generate random angle between 0 and 2*pi
    targetX = r * cos(angle);                                 // compute x coordinate
    targetY = r * sin(angle);                                 // compute y coordinate
    // targetX = 10;
    // targetY = 0;
    startingX = 0;
    startingY = 0;
    if (targetX == 0 && targetY == 0)
        targetX = 2;

    float optimal = hypotf(targetX, targetY) / 2.0 * hypotf(targetX, targetY);

    // transfer target coordinates to GPU

    startingParams[0] = targetX;
    startingParams[1] = targetY;
    startingParams[2] = optimal;
    startingParams[3] = startingX;
    startingParams[4] = startingY;
    startingParams[5] = iterationsCompleted;

    iterationsCompleted++;
}

/*
Two bots in this simulation. Calling them A and B.
The two bots control themselves via acceleration. They have 2 output neurons with (likely) linear activation.
This simulation is mainly to verify a single block can handle multiple bots.
Ax refers to A's x coordinate.
Avx is A's x velocity

Gamestate description
0 : iteration
1 : Ax
2 : Ay
3 : Avx
4 : Avy

5 : Bx
6 : By
7 : Bvx
8 : Bvy

11 : A_total_dist
12 : B_total_dist

13 : targetX
14 : targetY

15 : generation number

*/

// Right now both bots start in the same locaiton. Might change in future
__device__ void MultibotSimulation::setupSimulation(const float *startingParams, float *gamestate)
{
    if (threadIdx.x == 0)
    {
        // iter
        gamestate[0] = 0;

        // pos A
        gamestate[1] = startingParams[3];
        gamestate[2] = startingParams[4];

        // Vel A
        gamestate[3] = 0;
        gamestate[4] = 0;

        // pos B
        gamestate[5] = -startingParams[4];
        gamestate[6] = -startingParams[3];

        // Vel B
        gamestate[7] = 0;
        gamestate[8] = 0;

        // Distances
        gamestate[11] = 0;
        gamestate[12] = 0;

        // Target location
        gamestate[13] = startingParams[0];
        gamestate[14] = startingParams[1];

        gamestate[15] = startingParams[5]; //what generation we're on
    }
    __syncthreads();
}


__device__ unsigned int xorshift(unsigned int x)
{
    x ^= x >> 12;
    x ^= x << 25;
    x ^= x >> 27;
    return x * 0x2545F491;
}

// Used for cheap (fast) random numbers in setActivations. Random numbers help the model fit to more general information.
__device__ float rng(float a, float b, unsigned int seed)
{    
    unsigned int r = xorshift(seed);   
    static const float m = 4294967296.0f; // 2^32
    return a + (b - a) * (static_cast<float>(r) / m);
}

__device__ void MultibotSimulation::setActivations(float *gamestate, float **activs, int iter)
{
    int bot = -1;
    int tid = threadIdx.x;
    const int numBotVars = 4;

    if (tid < numBotVars)
    {
        bot = 0;
    }
    else if (tid < numBotVars * 2)
    {
        bot = 1;
    }
    // rand for adding noise to other bot's information

    float rand = 0;
    float randomMagnitude = 100.0f / logf(gamestate[15] + 2.0f) + 100.0f / logf((float)(iter + 2));
    
    if (bot == 0)
    {
        activs[bot][tid] = gamestate[tid + 1]; //+1 since iter is 0.

        rand = rng(-randomMagnitude, randomMagnitude, tid + iter + blockIdx.x ^ (int)gamestate[15]);        
        activs[bot][tid + numBotVars] = gamestate[tid + numBotVars + 1] + rand;
        if((int)gamestate[15] % 20 == 0)
            activs[bot][tid + numBotVars] = 0;
    }
    else if (bot == 1)
    {
        activs[bot][tid - numBotVars] = gamestate[tid + 1]; //+1 since iter is 0.

        rand = rng(-randomMagnitude, randomMagnitude, tid + iter + blockIdx.x ^ (int)gamestate[15]);
        activs[bot][tid] = gamestate[tid - numBotVars + 1] + rand;
        if((int)gamestate[15] % 20 == 0)
            activs[bot][tid] = 0;
    }

    if (tid < 2)
    {
        bot = tid;
        gamestate[0] = iter;

        // Input the target position
        activs[bot][8] = gamestate[13];
        activs[bot][9] = gamestate[14];
    }
    
    __syncthreads();
}

__device__ void MultibotSimulation::eval(float **actions, float *gamestate)
{
    const int numBotVars = 4;
    int tid = threadIdx.x;
    int velOffset = 3; // + 3 = 1 (iter) + 2 (pos indecies)
    int posOffset = 1;
    int bot = -1;

    if (tid < 2)
        bot = 0;
    else if (tid < 4)
        bot = 1;

    int direction = tid % 2; // which direction (x or y) this thread updates

    // // update velocities
    // if (tid < 4)
    // {
    //     // Allows precise movement in either direction.
    //     float preference = actions[bot][direction];
        
    //     //float accel = preference * MAX_ACCEL;

    //     // Bound the acceleration change
    //     //accel = fminf(MAX_ACCEL, fmaxf(-MAX_ACCEL, accel));

    //     // Update the bot's velocity
    //     //gamestate[bot * numBotVars + direction + velOffset] += accel;
    //     // gamestate[bot * numBotVars + direction + velOffset] = accel;
    // }



    __syncthreads();

    // update the bots' position
    if (tid < 2)
    {
        bot = tid;
        float accelX = actions[bot][0] * MAX_ACCEL;
        float accelY = actions[bot][1] * MAX_ACCEL;

        float accel = hypotf(accelX, accelY);
        if(accel > MAX_ACCEL){
            float f = MAX_ACCEL / accel;
            accelX *= f;
            accelY *= f;
        }
        
        gamestate[bot * numBotVars + velOffset + 0] += accelX;
        gamestate[bot * numBotVars + velOffset + 1] += accelY;
        float Avx = gamestate[bot * numBotVars + velOffset + 0];
        float Avy = gamestate[bot * numBotVars + velOffset + 1];

        // Make sure the speed doesn't go above max speed
        float speed = hypotf(Avx, Avy);
        if (speed > MAX_SPEED2)
        {
            float f = MAX_SPEED2 / speed;
            gamestate[bot * numBotVars + velOffset + 0] *= f;
            gamestate[bot * numBotVars + velOffset + 1] *= f;
        }

        // Update the bot's position
        gamestate[bot * numBotVars + posOffset + 0] += gamestate[bot * numBotVars + velOffset + 0];
        gamestate[bot * numBotVars + posOffset + 1] += gamestate[bot * numBotVars + velOffset + 1];
    }

    __syncthreads();
}

__device__ int MultibotSimulation::checkFinished(float *gamestate)
{
    int tid = threadIdx.x;
    if (tid < 2)
    {
        int bot = tid;
        int posOffset = 1;
        const int numBotVars = 4;

        float dx = gamestate[13] - gamestate[bot * numBotVars + posOffset + 0];
        float dy = gamestate[14] - gamestate[bot * numBotVars + posOffset + 1];
        float dist = hypotf(dx, dy);

        gamestate[bot + 11] += dist; // 11 = distOffset

        if (dist < 0.5f && threadIdx.x == 0)
        {
            //   printf("%d at target on iter %f\n", blockIdx.x, gamestate[0]);
        }

        // if (dist < .5f && threadIdx.x == 0 && gamestate[0] > 30)
        // {
        //     printf("dist = %f, iter = %d, bot = %d\n", dist, (int)gamestate[0], blockIdx.x);
        //     printf("Game state:\n");
        //     for(int i = 0; i < 15; i++)
        //         printf("%f, ", gamestate[i]);
        //     printf("\n");
        // }
    }
    __syncthreads();

    int resetInterval = 2500;
    // Check if we need to reset the sim this iteration
    if (((int)gamestate[0] + 1) % resetInterval == 0)
    {
        if (threadIdx.x == 0)
        {
            

            //"Rotate" the target position

            float new_x = gamestate[13] * cosf(ROTATION_ANGLE) - gamestate[14] * sinf(ROTATION_ANGLE);
            float new_y = gamestate[13] * sinf(ROTATION_ANGLE) + gamestate[14] * cosf(ROTATION_ANGLE);

            // Update the coordinates
            gamestate[13] = new_x;
            gamestate[14] = new_y;
        }
    }

    __syncthreads();

    // return dist < epsilon;
    return false;
}


__device__ void MultibotSimulation::setOutput(float *output, float *gamestate, const float * startingParams_d)
{   
    static int counter = 0;
    // output[block * 2] = (startingParams[2] / gamestate[11]); // Uses efficiency as a metric
    // output[block * 2 + 1] = (startingParams[2] / gamestate[12]); // Uses efficiency as a metric
                        

    if (threadIdx.x == 0)
    {
        if (gamestate[11] != 0)
            output[blockIdx.x * 2] = -gamestate[11];
        else
            output[blockIdx.x * 2] = 0;

        if (gamestate[12] != 0)
            output[blockIdx.x * 2 + 1] = -gamestate[12];
        else
            output[blockIdx.x * 2 + 1] = 0;

        if (blockIdx.x == 0)
        {
            if (counter % 25 == 0)
                printf("Block %d total dist = %f, efficiency = %f, counter = %d\n", blockIdx.x, gamestate[11], (startingParams_d[2] / gamestate[11]), counter);

            counter++;
        }
    }
}

__host__ int MultibotSimulation::getID()
{
    return 3;
}