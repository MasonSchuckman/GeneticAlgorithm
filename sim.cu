//#ifndef debug
//#define DEBUG 1
//#endif


#include "hip/hip_runtime.h"
#include "math.h"
#include <math.h>



#include <stdio.h>


void sharedTest();



//This is basically a #define. Needed for compile time understanding of certain variable definitions (particularly those that use numLayers for the size of an array)
constexpr auto numLayers = 3; //Number of layers;

constexpr auto bpt = 1; // Stands for "bots per thread" This allows for easy adjusting of the simulation if we wanna do something funky.

__host__ __device__ void printNet(float* activs, int* layerShapes) {
	printf("Activations:\n");
	int AO = 0; // "activs offset"

	for (int layer = 0; layer < numLayers; layer++) {
		printf("Layer %d, size = %d, AO = %d\n", layer, layerShapes[layer], AO);

		for (int i = 0; i < layerShapes[layer]; i++) {
			printf("%f, ", activs[AO + i]);
		}
		AO += layerShapes[layer];
		printf("\n");
	}
	printf("\n");
}


/**
 * Perform forward propagation of a dense neural network
 *
 * @param input     input data to the network, a float array of size input_size
 * @param weights   weight matrix of the network, a float array of size input_size * output_size
 * @param biases    bias vector of the network, a float array of size output_size
 * @param output    output of the network, a float array of size output_size
 * @param input_size    size of the input data
 * @param output_size   size of the output data
 */
__device__ void forward_propagation(const float* inputs, const float* weights, const float* biases, float* output, int input_size, int output_size)
{
	int stride = blockDim.x;
	int tid = threadIdx.x;
#ifdef DEBUG
	if (threadIdx.x == 0) {
		printf("Biases : ");
		for (int i = 0; i < output_size; i++) {
			printf("%f, ", biases[i]);
		}
		printf("\n");
	}
#endif
	// Initialize output to biases
	for (int i = threadIdx.x; i < output_size; i += stride) {
		output[i] = biases[i];
	}
	
	// Compute dot product of input and weights
#pragma unroll 4
	for (int i = 0; i < input_size; i++) {
		for (int j = tid; j < output_size; j += stride) {
			output[j] += inputs[i] * weights[i * output_size + j];
		}
	}


	

#ifdef DEBUG
	if (threadIdx.x == 0 && blockIdx.x == 0) {

		printf("Activs : ");
		for (int i = 0; i < output_size; i++)
			printf("%f, ", output[i]);
		printf("\n\n");
	}
#endif // DEBUG

	

	//TODO: Look into using different activation functions for different layers. (output should probably be sigmoid, others maybe ReLU)
	// Apply activation function (sigmoid in this case)	
	__syncthreads();
	for (int i = tid; i < output_size; i += stride) 
		output[i] = 1 / (1 + expf(-output[i]));
	
}


// this kernel divides the work into blocks rather than each thread works alone.
// Reason for doing this is to hopefully make better use of cache and reduce memory stalls.
// (Hopefully this will lead to higher FLOPS).
__global__ void simulateShared(const int n, const float* allWeights, const float* allBiases, const int* layerShapes, const float* startingParams, float* output) {

	int gid = threadIdx.x + blockIdx.x * blockDim.x; //global id
	int tid = threadIdx.x; //thread id (within a block)

	int block = blockIdx.x;
	int stride = blockDim.x;

	//prevent OOB errors
	if (block < n) {


		//calc the number of neurons per bot.
		int totalNeurons = 0;
		int totalWeights = 0;
		for (int i = 0; i < numLayers; i++) {
			if (i != numLayers - 1)
				totalWeights += layerShapes[i] * layerShapes[i + 1];
			totalNeurons += layerShapes[i];
		}

		//shared mem layout is w1,w2...,w_bpt,b1,b2,...,b_bpt,a_1,a_2,...,a_bpt
		//declare our block of shared memory
		extern __shared__ float s[];

		//split our shared memory block into the weights, biases, and activations
		float* weights = s;
		float* biases = weights + totalWeights * bpt;
		float* activations = biases + totalNeurons * bpt;

#ifdef DEBUG
		printf("Weights = %p\n", weights);
		printf("biases  = %p\n", biases);
		printf("activs  = %p\n", activations);
#endif

		//Copy this block's weights and biases to the shared arrays.
		for (int i = 0; i < totalWeights * bpt; i += stride) {
			weights[i] = (allWeights)[block * totalWeights + i];
		}
		for (int i = 0; i < totalNeurons * bpt; i += stride) {
			biases[i] = (allBiases)[block * totalNeurons + i];
		}


		//Seperate the bot(s) data
		const float* ws[bpt]; //abbreviation for "bot weights"
		const float* bs[bpt]; //abbreviation for "bot biases"
		float* activs[bpt]; //abbreviation for "bot activations"



		//Populate the arrays created above
		for (int i = 0; i < bpt; i++) {
			ws[i] = weights + totalWeights * i;
			bs[i] = biases + totalNeurons * i;
			activs[i] = activations + totalNeurons * i;
		}
		__syncthreads();

		int maxIters = 500;
		bool finished = false;

		int iter = 0; //current timestep of simulation we're on

		//run the simulation loop.
		while (!finished) {
			//Determine inputs for the bot(s)
			for (int i = 0; i < bpt; i++) {
				for (int j = tid; j < layerShapes[0]; j += stride) {
					//This line is a placeholder for now.
					activs[i][j] = 0.5f;
				}
			}


			//It's important to remember that activs and nns are essentially 2d arrays. That's why indexing them is tricky and weird.
			//Poll the NN for actions.
			for (int bot = 0; bot < bpt; bot++) {
				// All of these offsets are to account for the multiple layers in the network.
				int WO = 0; // weights offset
				int BO = 0; // biases offset
				int AO = 0; // activs offset
				int numBiases;
				int numWeights;
				for (int layer = 0; layer < numLayers - 1; layer++) {
					numBiases = layerShapes[layer];
					numWeights = numBiases * layerShapes[layer + 1];
#ifdef DEBUG
					if (tid == 0) {
						printf("Weights of layer %d:\n", layer);
						for (int k = 0; k < numBiases; k++) {
							for (int l = 0; l < layerShapes[layer + 1]; l++) {
								printf("%f, ", (nns[i] + LO + numBiases)[k * layerShapes[layer + 1] + l]);
							}
							printf("\n");
						}
						int cc = 0;
						for (int k = 0; k < numWeights; k++) {
							printf("%f, ", (nns[i] + LO + numBiases)[k]);
							cc++;
							if (cc % layerShapes[layer + 1] == 0)
								printf("\n");
						}
						printf("\n");
					}
					
#endif
					//forward_propagation(float* input, float* weights, float* biases, float* output, int input_size, int output_size)
					forward_propagation(activs[bot] + AO, ws[bot] + WO, bs[bot] + numBiases + BO, activs[bot] + AO + numBiases, numBiases, layerShapes[layer + 1]);

					AO += numBiases;
					WO += numWeights;
					BO += numBiases;
				}
			}

			//update simulation/game state based on bot actions




			//do simulation/game logic



			//if(checkWinCondition(<something>)
			//	finished = true;

			iter++;
			if (iter >= maxIters) {
				finished = true;
			}
		}
#ifdef DEBUG
		if (tid == 0 && blockIdx.x == 0) {
			printf("Activations:\n");
			int AO = 0; // "activs offset"
			for (int layer = 0; layer < numLayers; layer++) {
				printf("Layer %d, size = %d, AO = %d\n", layer, layerShapes[layer], AO);
				for (int i = 0; i < layerShapes[layer]; i++) {
					printf("%f, ", activs[0][AO + i]);
				}
				AO += layerShapes[layer];
				printf("\n");
			}
			printf("\n");
		}
#endif
	}
	return;
}


/*
* Note, I'll refer to the population as "bots"
* Brief description:
*   Using a dense neural network (NN)
*   There are N threads, corresponding to bpt * N bots. (these N's are lowercase in code)
*   networks memory layout in order:
		bpt * N * (b_ij + w_ij), where i=0...k, where k=numLayers is the number of layers
			and j=0...m, where m is the number of neurons in any layer.

			The effect of this memory layout is that each thread's NN data is contiguous in memory, rather than the data being
			interlaced.
*		Although the input layer doesn't have any biases, we will simply store them and have them set to 0 for code simplicity/cleanness
*
*   layerShapes is an array containing the size of each layer. There are numLayers layers.
*
*	The activations array is essentially scratch paper for all the intermediate steps while performing forward propagation.
*		NOTE: This scratch paper includes the inputs for each bot. This is to make the code for foward propagation cleaner.
*
*   Output will likely be an array of 2N length, with each entry corresponding to a bot's score. The host (CPU) will then
*       parse through the output and determine winners.
*
*/


//Defining the interface
using game_logic_func = int (*) (int);

//Actual function def here
__device__ int game1(int input){
	printf("in game\n");
	return 0;
}

// Required for functional pointer argument in kernel function
// Static pointers to device functions
__device__ game_logic_func game1_d = game1;



__global__ void game_kernel(game_logic_func game_logic) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    //output[tid] = game_logic(input[tid]);
	printf("%d\n", (*game_logic)(tid));	
	return;
}

void functionPointerTest(){


	hipSetDevice(0);
	game_logic_func game1_h;
	hipMemcpyFromSymbol(&game1_h, HIP_SYMBOL(game1_d), sizeof(game_logic_func));

	game_kernel<<<2, 32>>>(game1_h);
	hipDeviceSynchronize();

	printf("done\n");
}

int main()
{
	//functionPointerTest();

	sharedTest();


	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	
	return 0;
}

void sharedTest() {
	
	//int numThreads = 32;
	int totalBots =	1024 * 32;
	int numStartingParams = 1;
	int* layerShapes_h = new int[numLayers];
	float* startingParams_h = new float[numStartingParams];
	float* output_h = new float[totalBots];
	float* weights_h;
	float* biases_h;

	int numConnections = 0;
	int numNeurons = 0;

	// Determine the network configuration
	layerShapes_h[0] = 8;
	layerShapes_h[1] = 32;
	layerShapes_h[2] = 8;

	// Calculate how many connections and neurons there are based on layerShapes_h so we can create the networks_h array.
	for (int i = 0; i < numLayers; i++) {
		if(i != numLayers - 1)
			numConnections += layerShapes_h[i] * layerShapes_h[i + 1]; //effectively numWeights
		numNeurons += layerShapes_h[i];	//effectively numBiases
	}
	int botNetSize = (numConnections + numNeurons); //how many indices a single bot uses in the networks_h array.
	weights_h = new float[numConnections * totalBots];
	biases_h = new float[numNeurons * totalBots];


	printf("Total network size = %d KB\n", numConnections * sizeof(float) / (2 << 10));
	
	//initialize networks_h with random stuff for testing.

	for (int i = 0; i < totalBots; i++) {
		//printf("bot %d\n", i);
		int WO = 0;
		int BO = 0;
		for (int j = 0; j < numLayers; j++) {		
			//printf("\tlayer %d\n", j);
			for (int k = 0; k < layerShapes_h[j]; k++) {
				//printf("\t\tNode %d: ", k);
				//set the biases
				if (j == 0) {
					//input layer biases are 0
					biases_h[i * numNeurons + BO + k] = 0;
				}
				else {
					//other layers get a bias = layer number.
					biases_h[i * numNeurons + BO + k] = j;
				}
				//printf("bias = %f, weights: ", networks_h[i * botNetSize + LO + k]);
				if (j != numLayers - 1) {
					for (int l = 0; l < layerShapes_h[j + 1]; l++) {
						//set the weights. all layers get a weight of layerNum+1
						weights_h[i * numConnections + WO + k * layerShapes_h[j + 1] + l] = j + 1;
						//printf("%f, ", networks_h[i * botNetSize + LO + layerShapes_h[j] + k * layerShapes_h[j + 1] + l]);
					}
				}
				
				//printf("\n");
				
			}
			if (j != numLayers - 1) {
				BO += layerShapes_h[j];
				WO += layerShapes_h[j] * layerShapes_h[j + 1];
			}
			//printf("\n");
		}
		//printf("\n");
	}

	//printNet(networks_h, layerShapes_h); //this doesn't work I think

	//Create pointers for device data
	
	int* layerShapes_d;

	float* startingParams_d;
	float* output_d;
	float* weights_d;
	float* biases_d;


	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		//goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&layerShapes_d, numLayers * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		//goto Error;
	}

	hipMalloc((void**)&startingParams_d, numStartingParams * sizeof(float));
	hipMalloc((void**)&output_d, totalBots * sizeof(float));
	hipMalloc((void**)&weights_d, numConnections * totalBots * sizeof(float));
	hipMalloc((void**)&biases_d, totalBots * numNeurons * sizeof(float));


	//copy data over to GPU
	hipMemcpy(layerShapes_d, layerShapes_h, numLayers * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(startingParams_d, startingParams_h, numStartingParams * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(output_d, output_h, totalBots * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(weights_d, weights_h, totalBots * numConnections * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(biases_d, biases_h, totalBots * numNeurons * sizeof(float), hipMemcpyHostToDevice);



	int tpb = 32; //threads per block
	int numBlocks = (totalBots / bpt);
	printf("Num blocks = %d\n", numBlocks);

	int sharedMemNeeded = (numConnections + numNeurons * 2) * bpt;
	printf("Shared mem needed per block = %d KB\n", sharedMemNeeded * sizeof(float) / (2 << 10));
	// Launch a kernel on the GPU with one thread for each element.
	//__global__ void simulateShared(const int n, const float* allWeights, const float* allBiases, const int* layerShapes, const float* startingParams, float* output)
	simulateShared <<<numBlocks, tpb, sharedMemNeeded * sizeof(float)>>> (numBlocks, weights_d, biases_d, layerShapes_d, startingParams_d, output_d);


	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching simulate!\n", cudaStatus);
		goto Error;
	}


	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output_h, output_d, totalBots * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//clean up memory.
Error:
	hipFree(layerShapes_d);
	hipFree(startingParams_d);
	hipFree(output_d);
	hipFree(weights_d);
	hipFree(biases_d);

	
	delete[] layerShapes_h;
	delete[] startingParams_h;
	delete[] output_h;
	delete[] weights_h;
	delete[] biases_h;


}