#include "hip/hip_runtime.h"
#include "SimulationList.cuh"
#include "Kernels.cuh"


#include "Simulation.cuh"
#include "Simulator.cuh"
#include "TargetSimulation.cuh"

#include <iostream>
#include <vector>
#include <chrono>
#include <math.h>

#include <fstream>
#include "json.hpp"
using json = nlohmann::json;


using std::vector;

void getNetInfo(int &numConnections, int &numNeurons, std::vector<int> layerShapes)
{
    // Calculate how many connections and neurons there are based on layerShapes_h so we can create the networks_h array.
    for (int i = 0; i < layerShapes.size(); i++)
    {
        if (i != layerShapes.size() - 1)
            numConnections += layerShapes[i] * layerShapes[i + 1]; // effectively numWeights
        numNeurons += layerShapes[i];                                // effectively numBiases
    }
}

FullSimConfig setupSimulation(const std::string& filename) {
    std::ifstream file(filename);
    json configFile;

    // Parse the JSON file
    try {
        file >> configFile;
    } catch (const json::parse_error& e) {
        std::cerr << "Failed to parse config file " << filename << ": " << e.what() << std::endl;
        exit(1);
    }

    // Read the simulation type from the configFile
    std::string simType = configFile["simulation"].get<std::string>();
    Simulation * sim;
    if (simType == "TargetSimulation") {
        sim = new TargetSimulation;
    } else if (simType == "typeB") {
        //sim = TargetSimulation::TypeB;
    } else {
        std::cerr << "Unknown simulation type: " << simType << std::endl;
        exit(1);
    }

    // Read the neural net configuration from the configFile
    int numLayers = configFile["neural_net"]["num_layers"].get<int>();    
    std::vector<int> layerShapes = configFile["neural_net"]["layer_shapes"].get<std::vector<int>>();
    int numConnections = 0;
    int numNeurons = 0;
    getNetInfo(numConnections, numNeurons, layerShapes);

    // Read the rest of the simulation configuration from the config
    int botsPerSim = configFile["bots_per_sim"].get<int>();
    int maxIters = configFile["max_iters"].get<int>();

    //Note: the totalBots we put in the json is log_2 of what we simulate.
    int totalBots = configFile["total_bots"].get<int>();
    totalBots = (int) std::pow(2, totalBots);

    int numStartingParams = configFile["num_starting_params"].get<int>();
    int directContest = configFile["direct_contest"].get<int>();
    int botsPerTeam = configFile["bots_per_team"].get<int>();

    int generations = configFile["generations"].get<int>();

    float baseMutationRate = configFile["base_mutation_rate"].get<float>();
    float minMutationRate = configFile["min_mutation_rate"].get<float>();
    float mutationDecayRate = configFile["mutation_decay_rate"].get<float>();

    SimConfig config(numLayers, numNeurons, numConnections, botsPerSim, maxIters, numStartingParams, directContest, botsPerTeam);
    for(int i = 0; i < layerShapes.size(); i++)
        config.layerShapes[i] = layerShapes[i];

    // Create and return the SimConfig object
    return FullSimConfig(sim, config, totalBots, generations, baseMutationRate, minMutationRate, mutationDecayRate);
}

// Define constant GPU memory for the config of our simulation.
// Note: This CAN be set at runtime
__constant__ SimConfig config_d;

void launchKernel(Simulation *derived, SimConfig &config)
{

    hipSetDevice(0);

    // Create the sim on the GPU.
    Simulation **sim_d;
    hipMalloc(&sim_d, sizeof(Simulation **));

    // Copy the config over to GPU memory
    check(hipMemcpyToSymbol(HIP_SYMBOL(config_d), &config, sizeof(SimConfig)));

    Kernels::createDerived<<<1, 1>>>(sim_d, derived->getID());
    check(hipDeviceSynchronize());

    int n = 6;
    Kernels::game_kernel<<<2, 3>>>(n, sim_d);
    check(hipDeviceSynchronize());

    Kernels::delete_function<<<1, 1>>>(sim_d);
    check(hipDeviceSynchronize());

    hipFree(sim_d);

    printf("done\n");
    // Code to launch the CUDA kernel with the configured parameters and function pointer
}



void test_simulation_1()
{
    // Define which simulation we're running
    BasicSimulation sim;

    // Define the neural net configuration for our bots
    int numLayers = 3;
    int numConnections = 0, numNeurons = 0;
    vector<int> layerShapes;
    layerShapes.push_back(8);
    layerShapes.push_back(32);
    layerShapes.push_back(8);
      
    getNetInfo(numConnections, numNeurons, layerShapes);


    // Define the rest of the simulation configuration
    int botsPerSim = 1;

    if(botsPerSim > MAX_BOTS_PER_SIM){
        printf("Increase MAX_BOTS_PER_SIM and run again.\n");
        
    }

    int maxIters = 5;
    int totalBots = 1 << 15;
    int numStartingParams = 1;
    int directContest = 0;
    int botsPerTeam = 0;

    SimConfig config(numLayers, numNeurons, numConnections, botsPerSim, maxIters, numStartingParams, directContest, botsPerTeam);
    for (int i = 0; i < numLayers; i++)
    {
        config.layerShapes[i] = layerShapes[i];
    }
    
    vector<Bot*> bots;
    for(int i = 0; i < totalBots; i++){
        bots.push_back(new Bot(layerShapes));
    }

    printf("Created bots.\n");


    Simulator engine(bots, &sim, config);

    engine.batchSimulate(1);

    
    for(int i = 0; i < totalBots; i++){
        delete bots[i];
    }

    

}

void test_simulation_2()
{
    FullSimConfig fullConfig = setupSimulation("TargetSimConfig.json");

    vector<Bot*> bots;
    for(int i = 0; i < fullConfig.totalBots; i++){
        bots.push_back(new Bot(fullConfig.config.layerShapes, fullConfig.config.numLayers));
    }

    Simulator engine(bots, fullConfig.sim, fullConfig.config);
    engine.min_mutate_rate = fullConfig.minMutationRate;
    engine.mutateMagnitude = fullConfig.baseMutationRate;
    engine.mutateDecayRate = fullConfig.mutationDecayRate;
    engine.batchSimulate(fullConfig.generations);

    
    for(int i = 0; i < fullConfig.totalBots; i++){
        delete bots[i];
    }
}


int main()
{   
    hipSetDevice(0);

    auto start_time = std::chrono::high_resolution_clock::now();
    test_simulation_2();    
    auto end_time = std::chrono::high_resolution_clock::now();

    auto elapsed_time = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
    std::cout << "Total time taken: " << elapsed_time << " ms\n";

    // hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

    return 0;
}