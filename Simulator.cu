#include "hip/hip_runtime.h"
#include "Simulator.cuh"
#include <random>
#include <cmath>

using std::vector;

extern __constant__ SimConfig config_d;

// Constructor allocates all necessary device memory prior to doing simulations
Simulator::Simulator(vector<Specimen *> bots, Simulation *derived, SimConfig &config, Taxonomy *history) : bots{bots}, config{config}, derived{derived}, history{history}
{
    int totalBots = bots.size();

    int botNetSize = (config.totalNeurons + config.totalWeights); // how many indices a single bot uses in the networks_h array.

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    // Allocate GPU buffers
    // Note: all GPU arrays are member variables.
    cudaStatus = hipMalloc((void **)&layerShapes_d, config.numLayers * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
    }
    else
    {
        hipMalloc((void **)&startingParams_d, config.numStartingParams * sizeof(float));
        hipMalloc((void **)&output_d, totalBots * sizeof(float));

        hipMalloc((void **)&weights_d, config.totalWeights * totalBots * sizeof(float));
        hipMalloc((void **)&nextGenWeights_d, config.totalWeights * totalBots * sizeof(float));

        hipMalloc((void **)&biases_d, totalBots * config.totalNeurons * sizeof(float));
        hipMalloc((void **)&nextGenBiases_d, totalBots * config.totalNeurons * sizeof(float));

        hipMalloc((void **)&parentSpecimen_d, totalBots * sizeof(int));
        hipMalloc((void **)&distances_d, totalBots * sizeof(float));
        hipMalloc((void **)&ancestors_d, totalBots * sizeof(int));

        // Initialize as zeros
        hipMemset(distances_d, 0, totalBots * sizeof(float));

        int networkSize = (config.totalNeurons + config.totalWeights);
        // We need to pad deltas_d with zeros at the end of every bot's network so we can call reduce() on each bot's array easily.
        // To do that, each bot's deltas array needs to be a multiple of 32.
        int padding = 32 - (networkSize % 32);
        if (padding == 32)
            padding = 0;

        hipMalloc((void **)&deltas_d, totalBots * (networkSize + padding) * sizeof(float));
        hipMemset(deltas_d, 0, totalBots * (networkSize + padding) * sizeof(float));
        this->config.paddedNetworkSize = (networkSize + padding);

        // Copy the config over to GPU memory
        check(hipMemcpyToSymbol(HIP_SYMBOL(config_d), &this->config, sizeof(SimConfig)));

        // Setup the simulation class on the GPU
        hipMalloc(&sim_d, sizeof(Simulation **));
        Kernels::createDerived<<<1, 1>>>(sim_d, derived->getID());
        check(hipDeviceSynchronize());
    }
}

Simulator::~Simulator()
{
    hipFree(layerShapes_d);
    hipFree(startingParams_d);
    hipFree(output_d);
    hipFree(weights_d);
    hipFree(biases_d);
    hipFree(nextGenBiases_d);
    hipFree(nextGenWeights_d);
    hipFree(parentSpecimen_d);
    hipFree(distances_d);
    hipFree(deltas_d);
    hipFree(ancestors_d);

    // Free the simulation class on the GPU
    Kernels::delete_function<<<1, 1>>>(sim_d);
    check(hipDeviceSynchronize());

    hipFree(sim_d);
}

void Simulator::simulate()
{
    batchSimulate(1);
}

void Simulator::formatBotData(int *&layerShapes_h, float *&startingParams_h,
                              float *&output_h, float *&weights_h, float *&biases_h)
{
    for (int i = 0; i < config.numLayers; i++)
    {
        layerShapes_h[i] = config.layerShapes[i];
    }
    // for(int i = 0; i < config.numStartingParams; i++){
    //     startingParams_h[i] = config[i];
    // }

    int totalBots = bots.size();
    int i = 0;
    for (const Specimen *b : bots)
    {
        int WO = 0;
        int BO = 0;
        for (int j = 0; j < config.numLayers; j++)
        {
            for (int k = 0; k < layerShapes_h[j]; k++)
            {
                // set the biases
                if (j == 0)
                {
                    // input layer biases are 0
                    biases_h[i * config.totalNeurons + BO + k] = 0;
                }
                else
                {
                    // other layers get a bias = layer number.
                    biases_h[i * config.totalNeurons + BO + k] = 0;
                }
                if (j != config.numLayers - 1)
                {
                    for (int l = 0; l < layerShapes_h[j + 1]; l++)
                    {
                        // set the weights. all layers get a weight of layerNum+1
                        weights_h[i * config.totalNeurons + WO + k * layerShapes_h[j + 1] + l] = 0;
                    }
                }
            }
            if (j != config.numLayers - 1)
            {
                BO += layerShapes_h[j];
                WO += layerShapes_h[j] * layerShapes_h[j + 1];
            }
        }

        i++;
    }

    for (int i = 0; i < totalBots * config.totalNeurons; i++)
        biases_h[i] = 0;
    for (int i = 0; i < totalBots * config.totalWeights; i++)
        weights_h[i] = 0;
}

void Simulator::copyToGPU(int *&layerShapes_h, float *&startingParams_h,
                          float *&output_h, float *&weights_h, float *&biases_h)
{
    int totalBots = bots.size();
    check(hipMemcpy(layerShapes_d, layerShapes_h, config.numLayers * sizeof(int), hipMemcpyHostToDevice));
    check(hipMemcpy(startingParams_d, startingParams_h, config.numStartingParams * sizeof(float), hipMemcpyHostToDevice));
    check(hipMemcpy(output_d, output_h, totalBots * sizeof(float), hipMemcpyHostToDevice));
    check(hipMemcpy(weights_d, weights_h, totalBots * config.totalWeights * sizeof(float), hipMemcpyHostToDevice));
    check(hipMemcpy(nextGenWeights_d, weights_h, totalBots * config.totalWeights * sizeof(float), hipMemcpyHostToDevice));

    check(hipMemcpy(biases_d, biases_h, totalBots * config.totalNeurons * sizeof(float), hipMemcpyHostToDevice));
    check(hipMemcpy(nextGenBiases_d, biases_h, totalBots * config.totalNeurons * sizeof(float), hipMemcpyHostToDevice));

    // Quick and easy fix for initializing ancestors
    int *ancestors_h = new int[totalBots];
    for (int i = 0; i < totalBots; i++)
    {
        ancestors_h[i] = i;
    }
    check(hipMemcpy(ancestors_d, ancestors_h, totalBots * sizeof(int), hipMemcpyHostToDevice));
    delete[] ancestors_h;
}

// Copies the weights and biases of all the bots back to the host
void Simulator::copyFromGPU(float *&weights_h, float *&biases_h)
{

    int totalBots = bots.size();
    check(hipMemcpy(weights_h, nextGenWeights_d, totalBots * config.totalWeights * sizeof(float), hipMemcpyDeviceToHost));
    check(hipMemcpy(biases_h, nextGenBiases_d, totalBots * config.totalNeurons * sizeof(float), hipMemcpyDeviceToHost));
}

#include <fstream>
void writeWeightsAndBiasesAll(float *weights_h, float *biases_h, int TOTAL_BOTS, int totalWeights, int totalNeurons, int numLayers, int *layerShapes)
{
    std::ofstream outfile("allBots.data", std::ios::out | std::ios::binary); // this might be more space efficient
    // std::ofstream outfile("allBots.data");
    //  outfile << "all bots:\n";
    //  Write the total number of bots
    outfile.write(reinterpret_cast<const char *>(&TOTAL_BOTS), sizeof(int));

    // Write the total number of weights and neurons
    outfile.write(reinterpret_cast<const char *>(&totalWeights), sizeof(int));
    outfile.write(reinterpret_cast<const char *>(&totalNeurons), sizeof(int));

    // Write the number of layers and their shapes
    outfile.write(reinterpret_cast<const char *>(&numLayers), sizeof(int));
    for (int i = 0; i < numLayers; i++)
    {
        outfile.write(reinterpret_cast<const char *>(&layerShapes[i]), sizeof(int));
    }

    // Write the weights and biases for each bot
    for (int bot = 0; bot < TOTAL_BOTS; bot++)
    {
        // Write the weights for this bot
        for (int i = 0; i < totalWeights; i++)
        {
            float weight = weights_h[bot * totalWeights + i];
            outfile.write(reinterpret_cast<const char *>(&weight), sizeof(float));
        }

        // Write the biases for this bot
        int biasOffset = bot * totalNeurons;
        for (int i = 0; i < totalNeurons; i++)
        {
            float bias = biases_h[biasOffset + i];
            outfile.write(reinterpret_cast<const char *>(&bias), sizeof(float));
        }
    }

    outfile.close();
}

void write_weights_and_biases(float *weights, float *biases, int numLayers, int *layerShapes, int totalWeights, int totalNeurons, int lastGenBest)
{
    std::ofstream outfile("bestBot.data");
    outfile << "net_weights = np.array([";
    int WO = 0;
    for (int layer = 0; layer < numLayers - 1; layer++)
    {
        int numWeightsInLayer = layerShapes[layer] * layerShapes[layer + 1];
        outfile << "[";
        for (int i = 0; i < numWeightsInLayer; i++)
        {
            outfile << weights[lastGenBest * totalWeights + WO + i];
            if (i != numWeightsInLayer - 1)
                outfile << ", ";
        }
        WO += numWeightsInLayer;
        outfile << "]";
        if (layer != numLayers - 2)
            outfile << ",\n";
    }
    outfile << "])\n";

    int BO = layerShapes[0];
    outfile << "net_biases = np.array([";
    for (int layer = 1; layer < numLayers; layer++)
    {
        outfile << "[";
        for (int i = 0; i < layerShapes[layer]; i++)
        {
            outfile << biases[lastGenBest * totalNeurons + BO + i];
            if (i != layerShapes[layer] - 1)
                outfile << ", ";
        }
        BO += layerShapes[layer];
        outfile << "]";
        if (layer != numLayers - 1)
            outfile << ",\n";
    }
    outfile << "])\n";
    outfile.close();
}

void printError()
{
    printf("Error in loadData_()! Saved config doesn't match current config. Turn off load_data in the json.\n");
    exit(1);
}

// Dumb load. Assumes load will work (same number of bots and network config)
void Simulator::loadData_(float *weights_h, float *biases_h)
{
    std::ifstream infile("allBots.data", std::ios::in | std::ios::binary);
    if (!infile.is_open())
    {
        std::cerr << "Failed to open file\n";
        exit(1);
    }
    int placeholder;
    // Read the total number of bots
    infile.read(reinterpret_cast<char *>(&placeholder), sizeof(int));
    if (placeholder != bots.size())
    {
        printError();
    }

    // Read the total number of weights and neurons
    infile.read(reinterpret_cast<char *>(&placeholder), sizeof(int));
    if (placeholder != config.totalWeights)
    {
        printError();
    }
    infile.read(reinterpret_cast<char *>(&placeholder), sizeof(int));

    // Read the number of layers and their shapes
    infile.read(reinterpret_cast<char *>(&placeholder), sizeof(int));
    for (int i = 0; i < config.numLayers; i++)
        infile.read(reinterpret_cast<char *>(&placeholder), sizeof(int));

    int TOTAL_BOTS = bots.size();
    int totalWeights = config.totalWeights;
    int totalNeurons = config.totalNeurons;

    // Read the weights and biases for each bot
    for (int bot = 0; bot < TOTAL_BOTS; bot++)
    {
        // Read the weights for each layer
        for (int i = 0; i < totalWeights; i++)
        {
            float weight;
            infile.read(reinterpret_cast<char *>(&weight), sizeof(float));
            weights_h[bot * totalWeights + i] = weight;
        }

        // Read the biases for each layer
        for (int i = 0; i < totalNeurons; i++)
        {
            float bias;
            infile.read(reinterpret_cast<char *>(&bias), sizeof(float));
            biases_h[bot * totalNeurons + i] = bias;
        }
    }

    infile.close();
}

#include <sstream>
void Simulator::readWeightsAndBiasesAll(float *&weights_h, float *&biases_h, int &TOTAL_BOTS, int &totalWeights, int &totalNeurons, int &numLayers, int *layerShapes)
{
    std::ifstream infile("allBots.data", std::ios::in | std::ios::binary);
    if (!infile.is_open())
    {
        std::cerr << "Failed to open file\n";
        exit(1);
    }

    // Read the total number of bots
    infile.read(reinterpret_cast<char *>(&TOTAL_BOTS), sizeof(int));

    // Read the total number of weights and neurons
    infile.read(reinterpret_cast<char *>(&totalWeights), sizeof(int));
    infile.read(reinterpret_cast<char *>(&totalNeurons), sizeof(int));

    // Read the number of layers and their shapes
    infile.read(reinterpret_cast<char *>(&numLayers), sizeof(int));
    layerShapes = new int[numLayers];
    for (int i = 0; i < numLayers; i++)
    {
        infile.read(reinterpret_cast<char *>(&layerShapes[i]), sizeof(int));
    }

    // Allocate memory for the weights and biases
    weights_h = new float[TOTAL_BOTS * totalWeights];
    biases_h = new float[TOTAL_BOTS * totalNeurons];

    // Read the weights and biases for each bot
    for (int bot = 0; bot < TOTAL_BOTS; bot++)
    {
        // Read the weights for each layer
        for (int i = 0; i < totalWeights; i++)
        {
            float weight;
            infile.read(reinterpret_cast<char *>(&weight), sizeof(float));
            weights_h[bot * totalWeights + i] = weight;
        }

        // Read the biases for each layer
        for (int i = 0; i < totalNeurons; i++)
        {
            float bias;
            infile.read(reinterpret_cast<char *>(&bias), sizeof(float));
            biases_h[bot * totalNeurons + i] = bias;
        }
    }

    infile.close();
}

void read_weights_and_biases(float *weights, float *biases, int numLayers, int *layerShapes, int totalWeights, int totalNeurons, int lastGenBest)
{
    std::ifstream infile("bestBot.data");
    std::string line;
    std::vector<float> weights_vec;
    std::vector<float> biases_vec;
    bool reading_weights = false;
    bool reading_biases = false;
    int WO = 0;
    int BO = layerShapes[0];
    int layer = 0;

    while (std::getline(infile, line))
    {
        std::istringstream iss(line);
        std::string token;

        while (std::getline(iss, token, '['))
        {
            if (token.find("net_weights") != std::string::npos)
            {
                reading_weights = true;
                continue;
            }
            else if (token.find("net_biases") != std::string::npos)
            {
                reading_biases = true;
                continue;
            }
            if (!reading_weights && !reading_biases)
            {
                continue;
            }
            else if (reading_weights && token.find("]") != std::string::npos)
            {
                reading_weights = false;
                layer++;
            }
            else if (reading_biases && token.find("]") != std::string::npos)
            {
                reading_biases = false;
                layer++;
            }
            else
            {
                std::istringstream inner_iss(token);
                std::string inner_token;

                while (std::getline(inner_iss, inner_token, ','))
                {
                    if (reading_weights)
                    {
                        weights_vec.push_back(std::stof(inner_token));
                    }
                    else if (reading_biases)
                    {
                        biases_vec.push_back(std::stof(inner_token));
                    }
                }
            }
        }
    }

    int weight_index = lastGenBest * totalWeights;
    int bias_index = lastGenBest * totalNeurons;
    WO = 0;
    BO = layerShapes[0];

    for (int layer = 0; layer < numLayers - 1; layer++)
    {
        int numWeightsInLayer = layerShapes[layer] * layerShapes[layer + 1];

        for (int i = 0; i < numWeightsInLayer; i++)
        {
            weights[weight_index + WO + i] = weights_vec[WO + i];
        }
        WO += numWeightsInLayer;
    }

    for (int layer = 1; layer < numLayers; layer++)
    {
        int numBiasesInLayer = layerShapes[layer];

        for (int i = 0; i < numBiasesInLayer; i++)
        {
            biases[bias_index + BO + i] = biases_vec[BO + i];
        }
        BO += numBiasesInLayer;
    }

    infile.close();
}

#include <hipcub/hipcub.hpp>
float Simulator::getAvgDistance()
{
    // Allocate storage for the sum
    float *sum_d;
    hipMalloc((void **)&sum_d, 1 * sizeof(float));
    hipDeviceSynchronize();

    size_t temp_storage_bytes;
    int *temp_storage = NULL;
    hipcub::DeviceReduce::Sum(temp_storage, temp_storage_bytes, distances_d, sum_d, bots.size());
    hipMalloc(&temp_storage, temp_storage_bytes);
    hipDeviceSynchronize();

    hipcub::DeviceReduce::Sum(temp_storage, temp_storage_bytes, distances_d, sum_d, bots.size());
    hipDeviceSynchronize();
    // Copy result back to host
    float *sum_h = new float[1];
    check(hipMemcpy(sum_h, sum_d, 1 * sizeof(float), hipMemcpyDeviceToHost));
    // Get average
    *sum_h /= bots.size();

    // printf("Avg distance = %f\n", *sum_h);
    return *sum_h;
}

#include <chrono>
void Simulator::runSimulation(float *output_h, int *parentSpecimen_h, int *ancestors_h, float *distances_h)
{
    int printInterval = 25;

    int totalBots = bots.size();
    int tpb = 32; // threads per block
    int numBlocks = (totalBots / config.bpb);

    int sharedMemNeeded = (config.totalWeights + config.totalNeurons * 2) * config.bpb;
    if (iterationsCompleted == 0)
    {
        printf("Num blocks = %d. Bots per sim = %d\n", numBlocks, config.bpb);
        printf("Shared mem needed per block = %d KB\n", sharedMemNeeded * sizeof(float) / (2 << 10));
    }

    float *startingParams_h = new float[config.numStartingParams];
    derived->getStartingParams(startingParams_h);

    check(hipMemcpy(startingParams_d, startingParams_h, config.numStartingParams * sizeof(float), hipMemcpyHostToDevice));
    delete[] startingParams_h;

    auto start_time = std::chrono::high_resolution_clock::now();
    auto end_time = std::chrono::high_resolution_clock::now();
    auto elapsed_time = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();


    if (this->loadData == -1)
    {
        //printf("\nRunning shared mem version\t");

        start_time = std::chrono::high_resolution_clock::now();
        // Launch a kernel on the GPU with one block for each simulation/contest
        Kernels::simulateShared2<<<numBlocks, tpb, sharedMemNeeded * sizeof(float)>>>(numBlocks, this->sim_d, weights_d, biases_d, startingParams_d, output_d);

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        check(hipDeviceSynchronize());
        end_time = std::chrono::high_resolution_clock::now();
        elapsed_time = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
        if (iterationsCompleted % printInterval == 0)
            std::cout << "(Shared ver) Simulation time taken: " << elapsed_time << " ms\t";

        check(hipMemcpy(output_h, output_d, totalBots * sizeof(float), hipMemcpyDeviceToHost));
        float totalClocks = 0;
        for (int i = 0; i < totalBots; i++)
        {
            totalClocks += output_h[i];
        }
        //printf("Total clocks = %f\n", totalClocks);
    }
    else
    {

        //printf("Running constant mem version\t");

        start_time = std::chrono::high_resolution_clock::now();
        // Launch a kernel on the GPU with one block for each simulation/contest
        Kernels::simulateShared3<<<numBlocks, tpb, sharedMemNeeded * sizeof(float)>>>(numBlocks, this->sim_d, weights_d, biases_d, startingParams_d, output_d);

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        check(hipDeviceSynchronize());
        end_time = std::chrono::high_resolution_clock::now();
        elapsed_time = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
        if (iterationsCompleted % printInterval == 0)
            std::cout << "(Const ver) Simulation time taken: " << elapsed_time << " ms\t";

        check(hipMemcpy(output_h, output_d, totalBots * sizeof(float), hipMemcpyDeviceToHost));
        float totalClocks = 0;
        for (int i = 0; i < totalBots; i++)
        {
            totalClocks += output_h[i];
        }
        //printf("Total clocks = %f\n", totalClocks);
    }

    // slowly reduce the mutation rate until it hits a lower bound
    if (mutateMagnitude > min_mutate_rate)
        mutateMagnitude *= mutateDecayRate;

    // each block looks at 2 bots
    numBlocks = totalBots / 2; //(assumes even number of bots)
    // start_time = std::chrono::high_resolution_clock::now();

    int shift = (int)(((double)rand() / RAND_MAX) * totalBots * shiftEffectiveness) % totalBots;
    if (shiftEffectiveness < 0)
        shift = iterationsCompleted;

    float progThreshold = 1; // This will be calculated properly later

    auto start_time_mutate = std::chrono::high_resolution_clock::now();
    Kernels::mutate<<<numBlocks, tpb, config.paddedNetworkSize * sizeof(float)>>>(totalBots, mutateMagnitude, weights_d, biases_d, output_d, parentSpecimen_d,
                                                                                  nextGenWeights_d, nextGenBiases_d, distances_d, deltas_d, ancestors_d, progThreshold, iterationsCompleted, shift);

    check(hipDeviceSynchronize());
    end_time = std::chrono::high_resolution_clock::now();

    elapsed_time = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time_mutate).count();
    if (iterationsCompleted % printInterval == 0)
        std::cout << "Mutation time taken: " << elapsed_time << " ms\n";

    // swap which weights/biases arrays are "current"
    float *temp = nextGenBiases_d;
    nextGenBiases_d = biases_d;
    biases_d = temp;

    temp = nextGenWeights_d;
    nextGenWeights_d = weights_d;
    weights_d = temp;

    // Copy output vector from GPU buffer to host memory.
    check(hipMemcpy(output_h, output_d, totalBots * sizeof(float), hipMemcpyDeviceToHost));
    check(hipMemcpy(parentSpecimen_h, parentSpecimen_d, totalBots * sizeof(int), hipMemcpyDeviceToHost));
    check(hipMemcpy(ancestors_h, ancestors_d, totalBots * sizeof(int), hipMemcpyDeviceToHost));
    end_time = std::chrono::high_resolution_clock::now();
    // copy new generation from Device to Host

    // Used to decide where to write nextGen population data to
    iterationsCompleted++;
    if (iterationsCompleted % printInterval == 0)
    {
        elapsed_time = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
        printf("iter %d, mutate scale = %f. Shift = %d", iterationsCompleted, mutateMagnitude, shift);
        std::cout << " Generation took " << elapsed_time << " ms.\n";
    }
}

// void retrieveBotsToHost(float* weights_d, float* biases_d, vector<Bot*>* bots) {

//     check(hipMemcpy(&bots, output_d, bots.size() * sizeof(float), hipMemcpyDeviceToHost));
// }

void analyzeHistory(int numSimulations, int totalBots, float *output_h, int &finalBest)
{

    int printInterval = 25;
    int *bestIndexes = new int[numSimulations];
    float *bestScores = new float[numSimulations];
    float *averageScores = new float[numSimulations];

    // Loop over every generation
    for (int i = 0; i < numSimulations; i++)
    {
        float bestScore = -1000000;
        float averageScore = 0;
        int bestIndex = -1;

        // Find the best scoring bot of this generation
        for (int bot = 0; bot < totalBots; bot++)
        {
            float score = output_h[i * totalBots + bot];
            averageScore += score;
            if (score > bestScore)
            {
                bestScore = score;
                bestIndex = bot;
            }
        }

        // Record the results of this iteration
        bestIndexes[i] = bestIndex;
        bestScores[i] = bestScore;
        averageScores[i] = averageScore / totalBots;
    }

    // Summarize the results
    for (int i = 0; i < numSimulations; i += printInterval)
    {
        printf("Iteration : [%d]\tTop Score : %f, by Bot : [%d]\tAverage score : %f\n", i, bestScores[i], bestIndexes[i], averageScores[i]);
    }

    finalBest = bestIndexes[numSimulations - 1];
    // finalBest = 0;

    delete[] bestIndexes;
    delete[] bestScores;
    delete[] averageScores;
}

void printAncestry(Species *species, int offset)
{

    if (offset > 0)
    {
        std::cout << offset << "| ";
        for (int i = 0; i++ < offset; std::cout << "  ")
            ;
        std::cout << species->id << std::endl;
    }

    for (Species *subspecies : species->descendantSpecies)
        printAncestry(subspecies, offset + 1);
}

void historyGraph(Taxonomy *history)
{
    auto composition = history->speciesComposition();

    int lastRow = min((int)10, (int)composition->size());
    std::vector<std::tuple<Species *, float>> topCompositions(composition->begin(), composition->begin() + lastRow);

    for (int i = 0; i++ < 30; std::cout << std::endl)
        ;

    std::cout << "generation " << history->getYear() + 1 << std::endl;
    std::cout << history->compositionGraph(&topCompositions, 80) << std::endl;
    std::cout << Taxonomy::compositionString(&topCompositions) << std::endl
              << std::flush;
}

void Simulator::batchSimulate(int numSimulations)
{
    bool trackingGenetics = false;

    printf("num bots = %d, numLayers = %d, num weights = %d, numNeurons = %d\n", bots.size(), config.numLayers, config.totalWeights, config.totalNeurons);
    int totalBots = bots.size();

    // Allocate storage for bot data
    int *layerShapes_h = new int[config.numLayers];
    float *startingParams_h = new float[config.numStartingParams];
    float *output_h = new float[totalBots * numSimulations]; // We'll record all scores for all generations.
    float *weights_h = new float[config.totalWeights * totalBots];
    float *biases_h = new float[config.totalNeurons * totalBots];
    int *parentSpecimen_h = new int[totalBots];
    int *ancestors_h = new int[totalBots];
    float *distances_h = new float[totalBots];

    printf("Allocated host memory.\n");

    // Convert all the bot data to the format we need to transfer to GPU
    formatBotData(layerShapes_h, startingParams_h, output_h, weights_h, biases_h);

    printf("Formatted bot data.\n");

    if (loadData == 1)
    {
        loadData_(weights_h, biases_h);
        printf("Loaded in saved weights and biases.\n");
    }
    // Copy it over to the GPU
    copyToGPU(layerShapes_h, startingParams_h, output_h, weights_h, biases_h);

    printf("Copied data to GPU.\n");

    Specimen **previousGeneration;
    std::vector<std::vector<std::tuple<Species *, float>> *> compositions;
    if (trackingGenetics)
    {
        previousGeneration = new Specimen *[totalBots];
        for (int i = 0; i < totalBots; i++)
            previousGeneration[i] = bots.at(i);
    }

    // Invoke the kernel

    std::cout << "total variables in network (weights+biases): " << config.totalNeurons + config.totalWeights << std::endl;
    for (int i = 0; i < numSimulations; i++)
    {
        // Only pass the location to where this iteration is writing
        runSimulation(&output_h[i * totalBots], parentSpecimen_h, ancestors_h, distances_h);
        // build new speciment objects in order to log history
        copyFromGPU(weights_h, biases_h);

        if (trackingGenetics)
        {
            Specimen **nextGeneration = new Specimen *[totalBots];

            for (int j = 0; j < totalBots; j++)
            {
                Genome *nextGenome = new Genome(layerShapes_h, config.numLayers, &biases_h[j * config.totalNeurons], &weights_h[j * config.totalWeights], "sigmoid");
                Specimen *nextSpecimen = new Specimen(nextGenome, previousGeneration[parentSpecimen_h[j]]);

                nextGeneration[j] = nextSpecimen;
            }

            // bigger constant = harder to make a new species
            float MAGIC_CONSTANT = 50;
            float PROGENITOR_THRESHOLD = 0;

            PROGENITOR_THRESHOLD = getAvgDistance();
            PROGENITOR_THRESHOLD *= MAGIC_CONSTANT;

            history->incrementGeneration(nextGeneration, totalBots, PROGENITOR_THRESHOLD);
            compositions.push_back(history->speciesComposition());

            if (history->getYear() % 10 == 0)
                historyGraph(history);

            for (int j = 0; j < totalBots; j++)
                history->pruneSpecimen(previousGeneration[j]);

            delete previousGeneration;
            previousGeneration = nextGeneration;
            // printAncestry(previousGeneration[0]->species, 0);
        }
    }
    if (trackingGenetics)
        Taxonomy::writeCompositionsData(compositions, "comps.txt");
    printf("Ran simulation.\n");

    copyFromGPU(weights_h, biases_h);

    // Find the best score in each generation
    int lastGenBest = 0;
    analyzeHistory(numSimulations, totalBots, output_h, lastGenBest);

    write_weights_and_biases(weights_h, biases_h, config.numLayers, config.layerShapes, config.totalWeights, config.totalNeurons, lastGenBest);
    writeWeightsAndBiasesAll(weights_h, biases_h, totalBots, config.totalWeights, config.totalNeurons, config.numLayers, config.layerShapes);

    float *savedWeights;
    float *savedBiases;

    readWeightsAndBiasesAll(savedWeights, savedBiases, totalBots, config.totalWeights, config.totalNeurons, config.numLayers, config.layerShapes);

    int passed = 1;
    for (int i = 0; i < config.totalWeights * totalBots; i++)
    {
        if (savedWeights[i] != weights_h[i])
        {
            printf("iter %d\tsaved : %f\ttrue : %f\n", i, savedWeights[i], weights_h[i]);
            passed = 0;
        }
    }
    printf("PASSED TEST? %d\n", passed);

    delete[] savedWeights;
    delete[] savedBiases;

    // Do something with the output data....

    delete[] layerShapes_h;
    delete[] startingParams_h;
    delete[] output_h;
    delete[] weights_h;
    delete[] biases_h;
}
// Bot *Simulator::getBest()
// {
//     return nullptr;
// }
