#include "hip/hip_runtime.h"
#include "Simulator.cuh"
#include <random>
#include <cmath>

using std::vector;

extern __constant__ SimConfig config_d;

// Constructor allocates all necessary device memory prior to doing simulations
Simulator::Simulator(vector<Bot *> bots, Simulation *derived, SimConfig &config) : bots{bots}, config{config}, derived{derived}
{
    int totalBots = bots.size();

    int botNetSize = (config.totalNeurons + config.totalWeights); // how many indices a single bot uses in the networks_h array.

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    // Allocate GPU buffers
    // Note: all GPU arrays are member variables.
    cudaStatus = hipMalloc((void **)&layerShapes_d, config.numLayers * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
    }
    else
    {
        hipMalloc((void **)&startingParams_d, config.numStartingParams * sizeof(float));
        hipMalloc((void **)&output_d, totalBots * sizeof(float));

        hipMalloc((void **)&weights_d, config.totalWeights * totalBots * sizeof(float));
        hipMalloc((void **)&nextGenWeights_d, config.totalWeights * totalBots * sizeof(float));

        hipMalloc((void **)&biases_d, totalBots * config.totalNeurons * sizeof(float));
        hipMalloc((void **)&nextGenBiases_d, totalBots * config.totalNeurons * sizeof(float));

        // Copy the config over to GPU memory
        check(hipMemcpyToSymbol(HIP_SYMBOL(config_d), &config, sizeof(SimConfig)));

        // Setup the simulation class on the GPU
        hipMalloc(&sim_d, sizeof(Simulation **));
        Kernels::createDerived<<<1, 1>>>(sim_d, derived->getID());
        check(hipDeviceSynchronize());
    }
}

Simulator::~Simulator()
{
    hipFree(layerShapes_d);
    hipFree(startingParams_d);
    hipFree(output_d);
    hipFree(weights_d);
    hipFree(biases_d);
    hipFree(nextGenBiases_d);
    hipFree(nextGenWeights_d);

    // Free the simulation class on the GPU
    Kernels::delete_function<<<1, 1>>>(sim_d);
    check(hipDeviceSynchronize());

    hipFree(sim_d);
}

void Simulator::simulate()
{
    batchSimulate(1);
}

void Simulator::formatBotData(int *&layerShapes_h, float *&startingParams_h,
                              float *&output_h, float *&weights_h, float *&biases_h)
{
    for (int i = 0; i < config.numLayers; i++)
    {
        layerShapes_h[i] = config.layerShapes[i];
    }
    // for(int i = 0; i < config.numStartingParams; i++){
    //     startingParams_h[i] = config[i];
    // }

    int totalBots = bots.size();
    int i = 0;
    for (const Bot *b : bots)
    {
        int WO = 0;
        int BO = 0;
        for (int j = 0; j < config.numLayers; j++)
        {
            for (int k = 0; k < layerShapes_h[j]; k++)
            {
                // set the biases
                if (j == 0)
                {
                    // input layer biases are 0
                    biases_h[i * config.totalNeurons + BO + k] = 0;
                }
                else
                {
                    // other layers get a bias = layer number.
                    biases_h[i * config.totalNeurons + BO + k] = 0;
                }
                if (j != config.numLayers - 1)
                {
                    for (int l = 0; l < layerShapes_h[j + 1]; l++)
                    {
                        // set the weights. all layers get a weight of layerNum+1
                        weights_h[i * config.totalNeurons + WO + k * layerShapes_h[j + 1] + l] = 0;
                    }
                }
            }
            if (j != config.numLayers - 1)
            {
                BO += layerShapes_h[j];
                WO += layerShapes_h[j] * layerShapes_h[j + 1];
            }
        }

        i++;
    }

    for (int i = 0; i < totalBots * config.totalNeurons; i++)
        biases_h[i] = 0;
    for (int i = 0; i < totalBots * config.totalWeights; i++)
        weights_h[i] = 0;
}

void Simulator::copyToGPU(int *&layerShapes_h, float *&startingParams_h,
                          float *&output_h, float *&weights_h, float *&biases_h)
{
    int totalBots = bots.size();
    check(hipMemcpy(layerShapes_d, layerShapes_h, config.numLayers * sizeof(int), hipMemcpyHostToDevice));
    check(hipMemcpy(startingParams_d, startingParams_h, config.numStartingParams * sizeof(float), hipMemcpyHostToDevice));
    check(hipMemcpy(output_d, output_h, totalBots * sizeof(float), hipMemcpyHostToDevice));
    check(hipMemcpy(weights_d, weights_h, totalBots * config.totalWeights * sizeof(float), hipMemcpyHostToDevice));
    check(hipMemcpy(nextGenWeights_d, weights_h, totalBots * config.totalWeights * sizeof(float), hipMemcpyHostToDevice));

    check(hipMemcpy(biases_d, biases_h, totalBots * config.totalNeurons * sizeof(float), hipMemcpyHostToDevice));
    check(hipMemcpy(nextGenBiases_d, biases_h, totalBots * config.totalNeurons * sizeof(float), hipMemcpyHostToDevice));
}

// Copies the weights and biases of all the bots back to the host
void Simulator::copyFromGPU(float *&weights_h, float *&biases_h)
{

    int totalBots = bots.size();
    check(hipMemcpy(weights_h, nextGenWeights_d, totalBots * config.totalWeights * sizeof(float), hipMemcpyDeviceToHost));
    check(hipMemcpy(biases_h, nextGenBiases_d, totalBots * config.totalNeurons * sizeof(float), hipMemcpyDeviceToHost));
}

#include <chrono>

float mutateMagnitude = 1.0f; //starting magnitude
float min_mutate_rate = .0000001f; //ending magnitude
void Simulator::runSimulation(float *output_h)
{
    int totalBots = bots.size();
    int tpb = 32; // threads per block
    int numBlocks = (totalBots / config.bpb);
    // printf("Num blocks = %d\n", numBlocks);

    int sharedMemNeeded = (config.totalWeights + config.totalNeurons * 2) * config.bpb;
    // printf("Shared mem needed per block = %d KB\n", sharedMemNeeded * sizeof(float) / (2 << 10));

    // get random target coordinates
    int minPos = -config.maxIters * 2;
    int maxPos = config.maxIters * 2;
    std::random_device rd;                                 // obtain a random seed from hardware
    std::mt19937 eng(rd());                                // seed the generator
    std::uniform_int_distribution<> distr(minPos, maxPos); // define the range
    int targetX = distr(eng);
    int targetY = distr(eng);

    double r = 150.0; // radius of circle
    double angle = ((double)rand() / RAND_MAX) * 2 * 3.14159; // generate random angle between 0 and 2*pi
    // targetX = r * cos(angle); // compute x coordinate
    // targetY = r * sin(angle); // compute y coordinate

    if (targetX == 0 && targetY == 0)
        targetX = 2;
    
    float optimal = hypotf(targetX, targetY) / 2.0 * hypotf(targetX, targetY) * 4;

    // transfer target coordinates to GPU
    float *startingParams_h = new float[config.numStartingParams];
    
    startingParams_h[0] = targetX;
    startingParams_h[1] = targetY;
    startingParams_h[2] = optimal;

    check(hipMemcpy(startingParams_d, startingParams_h, config.numStartingParams * sizeof(float), hipMemcpyHostToDevice));
    delete[] startingParams_h;

    if (iterationsCompleted % 10 == 0)
    {
        printf("\nTarget at (%d, %d)\n", targetX, targetY);
        printf("Optimal dist = %f\n", optimal * 4);
    }
    auto start_time = std::chrono::high_resolution_clock::now();
    // Launch a kernel on the GPU with one block for each simulation/contest
    // Kernels::simulateShared<<<numBlocks, tpb, sharedMemNeeded * sizeof(float)>>>(numBlocks, this->sim_d, weights_d, biases_d, startingParams_d, output_d);
    // Kernels::simulateShared_noStaticArrays<<<numBlocks, tpb, sharedMemNeeded * sizeof(float)>>>(numBlocks, this->sim_d, weights_d, biases_d, startingParams_d, output_d);
    Kernels::simulateShared2<<<numBlocks, tpb, sharedMemNeeded * sizeof(float)>>>(numBlocks, this->sim_d, weights_d, biases_d, startingParams_d, output_d);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    check(hipDeviceSynchronize());
    auto end_time = std::chrono::high_resolution_clock::now();
    auto elapsed_time = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
    // std::cout << "Simulation time taken: " << elapsed_time << " ms\n";

    // Idea for mutating:
    /*
    if not a direct contest:
        One block looks at two bots, determined by the blockId * 2, and (blockId * 2 + iter*2 + 1 + (optional) shuffleSeed*2) % numBots
        The block can then write the 2 new children data where the parents were.

    if a direct contest:
        if 1 bot per team:
            each block looks at two bots, and checks which one won. The winner will split into two children, and the block will write the data to
            blockId * 2 and (blockId*2 + iter*2 + 1 + (optional) shuffleSeed*2) % numBots (assumes numBots is even.)
        if 2 bots per team:


    */

    
    // slowly reduce the mutation rate until it hits a lower bound
    if (mutateMagnitude > min_mutate_rate)
        mutateMagnitude *= 0.984f;

    // each block looks at 2 bots
    numBlocks = totalBots / 2; //(assumes even number of bots)
    //start_time = std::chrono::high_resolution_clock::now();
    // if(iterationsCompleted < 4)
    
    Kernels::mutate<<<numBlocks, tpb>>>(totalBots, mutateMagnitude, weights_d, biases_d, output_d, nextGenWeights_d, nextGenBiases_d, iterationsCompleted);
    check(hipDeviceSynchronize());
    end_time = std::chrono::high_resolution_clock::now();

    elapsed_time = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
    // std::cout << "Mutation time taken: " << elapsed_time << " ms\n";

    // swap which weights/biases arrays are "current"
    float *temp = nextGenBiases_d;
    nextGenBiases_d = biases_d;
    biases_d = temp;

    temp = nextGenWeights_d;
    nextGenWeights_d = weights_d;
    weights_d = temp;

    // Copy output vector from GPU buffer to host memory.
    check(hipMemcpy(output_h, output_d, totalBots * sizeof(float), hipMemcpyDeviceToHost));

    // Used to decide where to write nextGen population data to
    iterationsCompleted++;
    if (iterationsCompleted % 10 == 0){
        printf("iter %d, mutate scale = %f.", iterationsCompleted, mutateMagnitude);
        std::cout << " Generation took " << elapsed_time << " ms.\n";
    }
}

void analyzeHistory(int numSimulations, int totalBots, float *output_h, int &finalBest)
{

    int printInterval = 10;
    int *bestIndexes = new int[numSimulations];
    float *bestScores = new float[numSimulations];
    float *averageScores = new float[numSimulations];

    // Loop over every generation
    for (int i = 0; i < numSimulations; i++)
    {
        float bestScore = -1000000;
        float averageScore = 0;
        int bestIndex = -1;

        // Find the best scoring bot of this generation
        for (int bot = 0; bot < totalBots; bot++)
        {
            float score = output_h[i * totalBots + bot];
            averageScore += score;
            if (score > bestScore)
            {
                bestScore = score;
                bestIndex = bot;
            }
        }

        // Record the results of this iteration
        bestIndexes[i] = bestIndex;
        bestScores[i] = bestScore;
        averageScores[i] = averageScore / totalBots;
    }

    // Summarize the results
    for (int i = 0; i < numSimulations; i += printInterval)
    {
        printf("Iteration : [%d]\tTop Score : %f, by Bot : [%d]\tAverage score : %f\n", i, bestScores[i], bestIndexes[i], averageScores[i]);
    }

    finalBest = bestIndexes[numSimulations - 1];
    //finalBest = 0;

    delete[] bestIndexes;
    delete[] bestScores;
    delete[] averageScores;
}

void Simulator::batchSimulate(int numSimulations)
{

    printf("num bots = %d, numLayers = %d, num weights = %d, numNeurons = %d\n", bots.size(), config.numLayers, config.totalWeights, config.totalNeurons);
    int totalBots = bots.size();

    // Allocate storage for bot data
    int *layerShapes_h = new int[config.numLayers];
    float *startingParams_h = new float[config.numStartingParams];
    float *output_h = new float[totalBots * numSimulations]; // We'll record all scores for all generations.
    float *weights_h = new float[config.totalWeights * totalBots];
    float *biases_h = new float[config.totalNeurons * totalBots];
    printf("Allocated host memory.\n");

    // Convert all the bot data to the format we need to transfer to GPU
    formatBotData(layerShapes_h, startingParams_h, output_h, weights_h, biases_h);
    printf("Formatted bot data.\n");

    // Copy it over to the GPU
    copyToGPU(layerShapes_h, startingParams_h, output_h, weights_h, biases_h);
    printf("Copied data to GPU.\n");

    // Invoke the kernel
    for (int i = 0; i < numSimulations; i++)
    {
        // Only pass the location to where this iteration is writing
        runSimulation(&output_h[i * totalBots]);
    }
    printf("Ran simulation.\n");

    copyFromGPU(weights_h, biases_h);

    // Find the best score in each generation
    int lastGenBest = 0;
    analyzeHistory(numSimulations, totalBots, output_h, lastGenBest);

    // Print the last gen's best bot's weights and biases
    printf("net_weights = np.array([");
    int WO = 0;
    for (int layer = 0; layer < config.numLayers - 1; layer++)
    {
        int numWeightsInLayer = config.layerShapes[layer] * config.layerShapes[layer + 1];
        // printf("Layer %d, size = %d, WO = %d\n", layer, config.layerShapes[layer], WO);
        printf("[");
        for (int i = 0; i < numWeightsInLayer; i++)
        {
            printf("%f", weights_h[lastGenBest * config.totalWeights + WO + i]);
            if (i != numWeightsInLayer - 1)
                printf(", ");
        }
        WO += numWeightsInLayer;
        printf("]");
        if (layer != config.numLayers - 2)
            printf(",\n");
    }
    printf("])\n");


    printf("net_biases = np.array([");
    int BO = config.layerShapes[0];
    for (int layer = 1; layer < config.numLayers; layer++)
    {
        // printf("Layer %d, size = %d, WO = %d\n", layer, config.layerShapes[layer], WO);
        printf("[");
        for (int i = 0; i < config.layerShapes[layer]; i++)
        {
            printf("%f", biases_h[lastGenBest * config.totalNeurons + BO + i]);
            if (i != config.layerShapes[layer] - 1)
                printf(", ");
        }
        BO += config.layerShapes[layer];
        printf("]");
        if (layer != config.numLayers - 1)
            printf(",\n");
    }
    printf("])\n");
    

    // Do something with the output data....

    delete[] layerShapes_h;
    delete[] startingParams_h;
    delete[] output_h;
    delete[] weights_h;
    delete[] biases_h;
}

Bot *Simulator::getBest()
{
    return nullptr;
}
