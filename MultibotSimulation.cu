#include "hip/hip_runtime.h"
#include "MultibotSimulation.cuh"

// NOTE: this must be present in every derived simulation!
extern __constant__ SimConfig config_d;

__constant__ float MAX_SPEED2 = 5.0f;
__constant__ float MAX_ACCEL = 0.30f;

#define degrees 90.0f
#define ROTATION_ANGLE degrees * 3.141592654f / 180.0f // 90 degrees

/*
Two bots in this simulation. Calling them A and B.
The two bots control themselves via acceleration. They have 2 output neurons with (likely) linear activation.
This simulation is mainly to verify a single block can handle multiple bots.
Ax refers to A's x coordinate.
Avx is A's x velocity

Gamestate description
0 : iteration
1 : Ax
2 : Ay
3 : Bx
4 : By
5 : Avx
6 : Avy
7 : Bvx
8 : Bvy

11 : B_total_dist
12 : A_total_dist

13 : targetX
14 : targetY

*/

// Right now both bots start in the same locaiton. Might change in future
__device__ void MultibotSimulation::setupSimulation(const float *startingParams, float *gamestate)
{
    if (threadIdx.x == 0)
    {
        // iter
        gamestate[0] = 0;

        // pos A
        gamestate[1] = startingParams[3];
        gamestate[2] = startingParams[4];

        // pos B
        gamestate[3] = startingParams[3];
        gamestate[4] = startingParams[4];

        // Vel A
        gamestate[5] = 0;
        gamestate[6] = 0;

        // Vel B
        gamestate[7] = 0;
        gamestate[8] = 0;

        // Distances
        gamestate[11] = 0;
        gamestate[12] = 0;

        // Target location
        gamestate[13] = startingParams[0];
        gamestate[14] = startingParams[1];
    }
    __syncthreads();
}

__device__ void MultibotSimulation::setActivations(float *gamestate, float *activs, int iter)
{
    int tid = threadIdx.x;
    const int numInputs = 8; // not including target pos
    if (tid < numInputs)
    {
        activs[tid] = gamestate[tid + 1]; //+1 since iter is 0.
    }
    if (tid == 0)
    {
        gamestate[0] = iter;

        // Input the target position
        activs[8] = gamestate[13];
        activs[9] = gamestate[14];
    }
    __syncthreads();
}

__device__ void MultibotSimulation::eval(float **actions, float *gamestate)
{
    int tid = threadIdx.x;
    int bot = 0;
    if (tid < 2)
        bot = 0;
    else
        bot = 1;

    int posOffset = 1;
    int velOffset = 5;

    int direction = tid % 2; // which direction (x or y) this thread updates

    // update velocities
    if (tid < 4)
    {
        // Allows precise movement in either direction.
        float preference = actions[bot][direction];
        float accel = preference * MAX_ACCEL;

        // Bound the acceleration change
        accel = fminf(MAX_ACCEL, fmaxf(-MAX_ACCEL, accel));

        // Update the bot's velocity
        gamestate[bot * 2 + direction + velOffset] += accel;

       

        // if(gamestate[tid] > MAX_SPEED && blockIdx.x == 0){
        //     printf("ERROR IN EVAL. activation = %f\n", actions[0][tid]);
        // }
    }

    __syncthreads();

    // update the bots' position
    if (tid < 2)
    {
        bot = tid;

        float Avx = gamestate[bot * 2 + velOffset + 0];
        float Avy = gamestate[bot * 2 + velOffset + 1];

        // Make sure the speed doesn't go above max speed
        float speed = hypotf(Avx, Avy);
        if (speed > MAX_SPEED2)
        {
            float f = MAX_SPEED2 / speed;
            gamestate[bot * 2 + velOffset + 0] *= f;
            gamestate[bot * 2 + velOffset + 1] *= f;
        }

        // Update the bot's position
        gamestate[bot * 2 + posOffset + 0] += gamestate[bot * 2 + velOffset + 0];
        gamestate[bot * 2 + posOffset + 1] += gamestate[bot * 2 + velOffset + 1];
    }
    __syncthreads();
}

__device__ int MultibotSimulation::checkFinished(float *gamestate)
{
    int tid = threadIdx.x;
    if (tid < 2)
    {
        int bot = tid;
        int posOffset = 1;

        float dx = gamestate[13] - gamestate[bot * 2 + posOffset + 0];
        float dy = gamestate[14] - gamestate[bot * 2 + posOffset + 1];
        float dist = hypotf(dx, dy);

        gamestate[bot + 11] += dist; // 11 = distOffset

        if (dist < 0.5f && threadIdx.x == 0)
        {
            //   printf("%d at target on iter %f\n", blockIdx.x, gamestate[0]);
        }

        if (dist < .5f && threadIdx.x == 0 && gamestate[0] > 55)
        {
            //printf("dist = %f, iter = %d\n", dist, gamestate[0]);
        }

        __syncthreads();

        // Check if we need to reset the sim this iteration
        // if (((int)gamestate[7] + 1) % resetInterval == 0)
        // {
        //     if (threadIdx.x == 0)
        //     {
        //         // // Reset vel and pos
        //         // for(int i = 0; i < 4; i++)
        //         //     gamestate[i] = 0;

        //         //"Rotate" the target position

        //         float new_x = gamestate[4] * cosf(ROTATION_ANGLE) - gamestate[5] * sinf(ROTATION_ANGLE);
        //         float new_y = gamestate[4] * sinf(ROTATION_ANGLE) + gamestate[5] * cosf(ROTATION_ANGLE);

        //         // Update the coordinates
        //         gamestate[4] = new_x;
        //         gamestate[5] = new_y;
        //     }
        // }
    }

    __syncthreads();

    // return dist < epsilon;
    return false;
}

__host__ int MultibotSimulation::getID()
{
    return 3;
}